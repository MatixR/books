#include "hip/hip_runtime.h"
//  Single iteration of the Jacobi scheme (GPU kernel function).
//
//  (c) J. Blazek, CFD Consulting & Analysis, www.cfd-ca.de
//
//=============================================================================
//
//  This program is free software; you can redistribute it and/or
//  modify it under the terms of the GNU General Public License
//  as published by the Free Software Foundation; either version 2
//  of the License, or (at your option) any later version.
//
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License
//  along with this program; if not, write to the Free Software
//  Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA.
//
//*****************************************************************************

#include "defs.h"

// Basic kernel function which uses global memory only
//
__global__ void Jacobi( int Ni, int Nj, REAL *phi, REAL *phiOld )
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;  // global index

  if (i>0 && i<(Ni-1))
  {
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (j>0 && j<(Nj-1))
    {
      int ij  = i + j*Ni;
      phi[ij] = 0.25f*(phiOld[ij+1]+phiOld[ij-1]+phiOld[ij+Ni]+phiOld[ij-Ni]);
    }
  }
}

//*****************************************************************************

// Optimized kernel function which uses shared memory
//
__global__ void Jacobi_optim( int Ni, int Nj, int nThreadsX, int nThreadsY,
                              REAL *phi, REAL *phiOld )
{
  int i, j, ij, iLocal, jLocal, iLocalDum, jLocalDum;

  __shared__ REAL sharedMem[34][18];  // shared memory block including surrounding
                                      // dummy layer (1 deep) to cover the stencil
                                      // - sized as nThreadsX+2, nThreadsY+2

  i  = blockDim.x * blockIdx.x + threadIdx.x;  // global indexes
  j  = blockDim.y * blockIdx.y + threadIdx.y;
  ij = i + j*Ni;

  iLocal    = threadIdx.x;  // local indexes within grid block
  jLocal    = threadIdx.y;
  iLocalDum = iLocal + 1;   // including offset for dummy layer
  jLocalDum = jLocal + 1;

  if (iLocal == 0)  // copy left & right dummy layer (except at boundaries)
  {
    if (blockIdx.x > 0)
      sharedMem[iLocal][jLocalDum] = phiOld[ij-1];

	if (blockIdx.x < (gridDim.x-1))
      sharedMem[iLocalDum+nThreadsX][jLocalDum] = phiOld[ij+nThreadsX];
  }
  if (jLocal == 0)  // copy bottom & top dummy layer (except at boundaries)
  {
    if (blockIdx.y > 0)
	  sharedMem[iLocalDum][jLocal] = phiOld[ij-Ni];

	if (blockIdx.y < (gridDim.y-1))
      sharedMem[iLocalDum][jLocalDum+nThreadsY] = phiOld[ij+Ni*nThreadsY];
  }

  if (i<Ni && j<Nj)
    sharedMem[iLocalDum][jLocalDum] = phiOld[ij];  // copy interior elements

  __syncthreads();  // wait for all threads of the block to finish

  if (i>0 && i<(Ni-1))
  {
    if (j>0 && j<(Nj-1))
    {
      phi[ij] = 0.25f*(sharedMem[iLocalDum+1][jLocalDum]+sharedMem[iLocalDum-1][jLocalDum]+
                       sharedMem[iLocalDum][jLocalDum+1]+sharedMem[iLocalDum][jLocalDum-1]);
    }
  }
}
