#include "hip/hip_runtime.h"
//  Solution of the Laplace equation by Jacobi iteration on
//  a uniform structured grid with Ni x Nj nodes. X-coordinate
//  is associated with the i-direction, y-coordinate with j.
//  This example employs Nvidia's CUDA for the parallelization.
//
//  (c) J. Blazek, CFD Consulting & Analysis, www.cfd-ca.de
//
//=============================================================================
//
//  This program is free software; you can redistribute it and/or
//  modify it under the terms of the GNU General Public License
//  as published by the Free Software Foundation; either version 2
//  of the License, or (at your option) any later version.
//
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License
//  along with this program; if not, write to the Free Software
//  Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA.
//
//*****************************************************************************

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "defs.h"

int main( int argc, char** argv )
{
  int Ni = 512;        // number of nodes in i-(x-)direction
  int Nj = 512;        // number of nodes in j-(y-)direction
  int maxIter = 10000; // max. number of iterations
  int nThreadsX = 32;  // number of threads in x-direction
  int nThreadsY = 16;  // number of threads in y-direction

  REAL *phi=NULL;                     // solution on the host
  REAL *phiDev=NULL, *phiDevOld=NULL; // current and previous solution on the GPU

  float       elapsedTime;
  int         i, j;
  hipEvent_t start, stop;
  FILE        *fp;

  // parse the command line

  ParseCommandLine( argc,argv,&Ni,&Nj,&maxIter,&nThreadsX,&nThreadsY );
  printf("\nNi=%d, Nj=%d, maxIter=%d, nThreadsX=%d, nThreadsY=%d\n",Ni,Nj,maxIter,
         nThreadsX,nThreadsY);

  // memory for the result (on CPU)

  phi = (REAL*) calloc( Ni*Nj,sizeof(REAL) );

  // set boundary conditions at j=0, j=Nj-1

  for (i=0; i<Ni; i++)
  {
    phi[i+Ni* 0    ] = 5.0f;
    phi[i+Ni*(Nj-1)] = 5.0f;
  }

  // set boundary conditions at i=0, i=Ni-1

  for (j=1; j<(Nj-1); j++)
  {
    phi[0   +Ni*j] = 5.0f;
    phi[Ni-1+Ni*j] = 5.0f;
  }

  // initialize GPU stuff

  hipError_t cudaStatus = Initialize( Ni,Nj,phi,&phiDev,&phiDevOld );
  if (cudaStatus != hipSuccess)
  {
    printf("Initialization failed!\n");
    free( phi );
    Uninitialize( &phiDev,&phiDevOld );
    return 1;
  }

  dim3 threadsPerBlock( nThreadsX,nThreadsY );
  dim3 blocksPerGrid( (Ni+threadsPerBlock.x-1)/threadsPerBlock.x,
                      (Nj+threadsPerBlock.y-1)/threadsPerBlock.y );

  // perform iterations

  printf("Performing iterations ...\n");
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord( start,NULL );

  for (int iter=1; iter<=maxIter; iter++)
  {
    //Jacobi<<<blocksPerGrid,threadsPerBlock>>>( Ni,Nj,phiDev,phiDevOld );

    Jacobi_optim<<<blocksPerGrid,threadsPerBlock>>>( Ni,Nj,nThreadsX,nThreadsY,
                                                     phiDev,phiDevOld );

    REAL *tmp = phiDevOld;
    phiDevOld = phiDev;
    phiDev    = tmp;
  }

  hipEventRecord( stop,NULL );
  hipEventSynchronize( stop );
  hipEventElapsedTime( &elapsedTime,start,stop );
  printf("GPU Time: %g sec", elapsedTime/1000.0f);

  // copy solution to host's memory

  cudaStatus = hipMemcpy( phi,phiDev,(Ni*Nj)*sizeof(REAL),hipMemcpyDeviceToHost );
  if (cudaStatus != hipSuccess)
  {
    printf("hipMemcpy from PHIDEV to PHI failed!\n");
    return cudaStatus;
  }

  // free memory and reset GPU

  Uninitialize( &phiDev,&phiDevOld );

  // store solution in a plot file

  fp = fopen("plot.v2d","wt");
  fprintf(fp,"Solution of Laplace Equation\n");
  fprintf(fp,"1\n");
  fprintf(fp,"Field Plot\n");
  fprintf(fp,"1 3\n");
  fprintf(fp,"x\ny\nf\n");
  fprintf(fp,"%i %i\n",Ni,Nj);
  fprintf(fp,"0 0 0\n");
  fprintf(fp,"Structured\n");
  for (j=0; j<Nj; j++)
  {
    for (i=0; i<Ni; i++)
    {
      fprintf(fp,"%d %d %14.6e\n",i,j,phi[i+Ni*j]);
    }
  }
  fclose(fp);

  // finish

  free( phi );
  printf("\nFinished!\n");
  return EXIT_SUCCESS;
}

//*****************************************************************************

// Parses the command line
//
void ParseCommandLine( int argc, char**argv, int *Ni, int *Nj,
                       int *maxIter, int *nThreadsX, int *nThreadsY )
{
  if (argc >= 1)
  {
    for (int i=1; i<argc; i++)
    {
      int bFirstArgIsParam = false;
      int stringStart = 0;

      while (argv[i][stringStart] == '-') stringStart++;
      char *stringArgv = &argv[i][stringStart];

      if (!STRNCASECMP( stringArgv,"Ni=",3 ))
      {
        bFirstArgIsParam = true;
        *Ni = atoi(&stringArgv[3]);
        continue;
      }
      if (!STRNCASECMP( stringArgv,"Nj=",3 ))
      {
        bFirstArgIsParam = true;
        *Nj = atoi(&stringArgv[3]);
        continue;
      }
      if (!STRNCASECMP( stringArgv,"maxIter=",8 ))
      {
        bFirstArgIsParam = true;
        *maxIter = atoi(&stringArgv[8]);
        continue;
      }
      if (!STRNCASECMP( stringArgv,"nThreadsX=",10 ))
      {
        bFirstArgIsParam = true;
        *nThreadsX = atoi(&stringArgv[10]);
        continue;
      }
      if (!STRNCASECMP( stringArgv,"nThreadsY=",10 ))
      {
        bFirstArgIsParam = true;
        *nThreadsY = atoi(&stringArgv[10]);
        continue;
      }
      if (!bFirstArgIsParam || !STRNCASECMP( stringArgv,"h",1 ))
      {
        printf("Usage:\n\n"
               "jacobi [-h] [-Ni=] [-Nj=] [-maxIter=] [-nThreadsX=] [-nThreadsY=]\n");
        exit(0);
      }
    }
  }
}

//*****************************************************************************

// Initializes CUDA device and memory
//
hipError_t Initialize( int Ni, int Nj, REAL *phiHost, REAL **phi, REAL **phiOld )
{
  hipError_t cudaStatus;
  unsigned int size = Ni * Nj * sizeof(REAL);

  // choose which GPU to run on, change this on a multi-GPU system
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess)
  {
    printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
    return cudaStatus;
  }

  // allocate GPU buffers
  cudaStatus = hipMalloc( (void**)&(*phi),size );
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMalloc for PHI failed!\n");
    return cudaStatus;
  }

  cudaStatus = hipMalloc( (void**)&(*phiOld),size );
  if (cudaStatus != hipSuccess)
  {
    printf("hipMalloc for PHIOLD failed!\n");
    return cudaStatus;
  }

  // initialize GPU buffers

  cudaStatus = hipMemcpy( *phi,phiHost,size,hipMemcpyHostToDevice );
  if (cudaStatus != hipSuccess)
  {
    printf("hipMemcpy from PHIHOST to PHI failed!\n");
    return cudaStatus;
  }

  cudaStatus = hipMemcpy( *phiOld,phiHost,size,hipMemcpyHostToDevice );
  if (cudaStatus != hipSuccess)
  {
    printf("hipMemcpy from PHIHOST to PHIOLD failed!\n");
    return cudaStatus;
  }

  //hipDeviceSetSharedMemConfig( hipSharedMemBankSizeEightByte );
  //hipDeviceSetCacheConfig( hipFuncCachePreferShared );
  
  //hipComputeModeExclusive

  return hipSuccess;
}

//*****************************************************************************

// Cleans up CUDA device and memory
//
void Uninitialize( REAL **phi, REAL **phiOld )
{
  hipFree( *phi );
  hipFree( *phiOld );

  hipError_t cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess) printf("hipDeviceReset failed!\n");
}
